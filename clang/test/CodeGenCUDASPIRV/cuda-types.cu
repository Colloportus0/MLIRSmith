
#include <hip/hip_runtime.h>
// Check that types, widths, __CLANG_ATOMIC* macros, etc. match on the host and
// device sides of CUDA compilations. Note that we filter out long double and
// maxwidth of _BitInt(), as this is intentionally different on host and device.
//
// Also ignore __CLANG_ATOMIC_LLONG_LOCK_FREE on i386. The default host CPU for
// an i386 triple is typically at least an i586, which has cmpxchg8b (Clang
// feature, "cx8"). Therefore, __CLANG_ATOMIC_LLONG_LOCK_FREE is 2 on the host,
// but the value should be 1 for the device.
//
// Unlike CUDA, the width of SPIR-V POINTER type could differ between host and
// device, because SPIR-V explicitly sets POINTER type width. So it is the
// user's responsibility to choose the offload with the right POINTER size,
// otherwise the values for __CLANG_ATOMIC_POINTER_LOCK_FREE could be different.

// RUN: mkdir -p %t

// RUN: %clang --cuda-host-only -nocudainc -nocudalib --offload=spirv32 -target i386-unknown-linux-gnu -x cuda -emit-llvm -E -dM -o - /dev/null \
// RUN:   | grep -E '__CLANG_ATOMIC' \
// RUN:   | grep -Ev '_ATOMIC_LLONG_LOCK_FREE' > %t/i386-host-defines-filtered
// RUN: %clang --cuda-device-only -nocudainc -nocudalib --offload=spirv32 -target i386-unknown-linux-gnu -x cuda -emit-llvm -E -dM -o - /dev/null \
// RUN:   | grep -E '__CLANG_ATOMIC' \
// RUN:   | grep -Ev '_ATOMIC_LLONG_LOCK_FREE' > %t/i386-device-defines-filtered
// RUN: diff %t/i386-host-defines-filtered %t/i386-device-defines-filtered

// RUN: %clang --cuda-host-only -nocudainc -nocudalib --offload=spirv32 -target i386-windows-msvc -x cuda -emit-llvm -E -dM -o - /dev/null \
// RUN:   | grep -E '__CLANG_ATOMIC' \
// RUN:   | grep -Ev '_ATOMIC_LLONG_LOCK_FREE' > %t/i386-msvc-host-defines-filtered
// RUN: %clang --cuda-device-only -nocudainc -nocudalib --offload=spirv32 -target i386-windows-msvc -x cuda -emit-llvm -E -dM -o - /dev/null \
// RUN:   | grep -E '__CLANG_ATOMIC' \
// RUN:   | grep -Ev '_ATOMIC_LLONG_LOCK_FREE' > %t/i386-msvc-device-defines-filtered
// RUN: diff %t/i386-msvc-host-defines-filtered %t/i386-msvc-device-defines-filtered

// RUN: %clang --cuda-host-only -nocudainc -nocudalib --offload=spirv64 -target x86_64-unknown-linux-gnu -x cuda -emit-llvm -E -dM -o - /dev/null \
// RUN:   | grep -E '__CLANG_ATOMIC' \
// RUN:   | grep -Ev '_ATOMIC_LLONG_LOCK_FREE' > %t/x86_64-host-defines-filtered
// RUN: %clang --cuda-device-only -nocudainc -nocudalib --offload=spirv64 -target x86_64-unknown-linux-gnu -x cuda -emit-llvm -E -dM -o - /dev/null \
// RUN:   | grep -E '__CLANG_ATOMIC' \
// RUN:   | grep -Ev '_ATOMIC_LLONG_LOCK_FREE' > %t/x86_64-device-defines-filtered
// RUN: diff %t/x86_64-host-defines-filtered %t/x86_64-device-defines-filtered

// RUN: %clang --cuda-host-only -nocudainc -nocudalib --offload=spirv64 -target powerpc64-unknown-linux-gnu -x cuda -emit-llvm -E -dM -o - /dev/null \
// RUN:   | grep -E '__CLANG_ATOMIC' \
// RUN:   | grep -Ev '_ATOMIC_LLONG_LOCK_FREE' > %t/powerpc64-host-defines-filtered
// RUN: %clang --cuda-device-only -nocudainc -nocudalib --offload=spirv64 -target powerpc64-unknown-linux-gnu -x cuda -emit-llvm -E -dM -o - /dev/null \
// RUN:   | grep -E '__CLANG_ATOMIC' \
// RUN:   | grep -Ev '_ATOMIC_LLONG_LOCK_FREE' > %t/powerpc64-device-defines-filtered
// RUN: diff %t/powerpc64-host-defines-filtered %t/powerpc64-device-defines-filtered

// RUN: %clang --cuda-host-only -nocudainc -nocudalib --offload=spirv64 -target x86_64-windows-msvc -x cuda -emit-llvm -E -dM -o - /dev/null \
// RUN:   | grep -E '__CLANG_ATOMIC' \
// RUN:   | grep -Ev '_ATOMIC_LLONG_LOCK_FREE' > %t/x86_64-msvc-host-defines-filtered
// RUN: %clang --cuda-device-only -nocudainc -nocudalib --offload=spirv64 -target x86_64-windows-msvc -x cuda -emit-llvm -E -dM -o - /dev/null \
// RUN:   | grep -E '__CLANG_ATOMIC' \
// RUN:   | grep -Ev '_ATOMIC_LLONG_LOCK_FREE' > %t/x86_64-msvc-device-defines-filtered
// RUN: diff %t/x86_64-msvc-host-defines-filtered %t/x86_64-msvc-device-defines-filtered

